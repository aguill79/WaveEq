#include "hip/hip_runtime.h"
// Example physical wave program
//   --a medium is bounded by a square box that act as reflective boundary
//   --conditions. A pulse is made in the middle of this box
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

//-------------------------------------------------------------------------
// Included Portable timer functions 
//-------------------------------------------------------------------------
#include "timer.h"

#define TILE_WIDTH 16 // block x and y dimensions

// iceil macro
// returns an integer ceil value where integer numerator is first parameter
// and integer denominator is the second parameter. iceil is the rounded
// up value of numerator/denominator when there is a remainder
#define iceil(num,den) (num/den+(num%den>0))


__global__ void WaveKernel(float *unew, float *u, float *uold, int NN,
                           float rho2) {

   // Calculate the row index of the Pd element and M
   int Row = blockIdx.y*TILE_WIDTH+threadIdx.y + 1;
   int Col = blockIdx.x*TILE_WIDTH+threadIdx.x + 1;

   if ((Row < NN-1) && (Col < NN-1)) {
     unew[Row*NN+Col]=2.0*(1-2.0*rho2)*u[Row*NN+Col] + rho2*(u[(Row+1)*NN+Col]+ 
                      u[(Row-1)*NN+Col] + u[Row*NN+(Col-1)] + u[Row*NN+(Col+1)])
                     - uold[Row*NN+Col];
   }
}

// gnu_plot compatable output
void output_wave_gnu(int NN, float *u, char *file) {
   int i,j;
   char fname[30];
   FILE *fp;
   sprintf(fname,"%s.gdt",file); // create gnuplot data file
   if ((fp = fopen(fname,"w")) == NULL) {
      perror("Unable to open gnuplot output file");
      exit(1);
   };

   for (i=1; i<NN-1; i++) {
      for (j=1; j<NN-1; j++) {
         fprintf(fp,"%d %d %f\n",i,j,u[i*NN+j]);
      }
      fprintf(fp,"\n");
   }
   fclose(fp);
}

// pdmdump compatable output
void pbmdump(int N, float *u, char * file) {

   float min, max, range;
   int i,j;
   FILE  *fp;

   min=max=u[0];
   for (i=0; i<=N; i++) 
      for (j=0; j<=N; j++) {
         if (min > u[i*N+j]) min=u[i*N+j];
         if (max < u[i*N+j]) max=u[i*N+j];
      }
   range = max-min;

   if ((fp = fopen(file,"wb")) == NULL) {
      perror("Unable to open PBM dump file");
   } else {
      fprintf(fp,"P5\n%d %d\n255\n",N,N);
      for (i=0; i<N; i++) 
         for (j=0; j<N; j++) 
            fputc((char) (((u[i*N+j]-min)/range)*255.0), fp);
      fclose(fp);
   }
}


int main(int argc, char *argv[]) {

   char  file[32] = "dump";
   float *u=NULL,*uold=NULL,*tmp_ptr=NULL;
   float rho,rho2,h,dt;
   int i,j,t,size;
   int N,NN,steps;
   hipError_t error_id;
   double timer_val;


   if (argc < 3) {
      perror("Usage: seq_wave <size> <steps> [<output file>] ");
      exit(1);
   }
   sscanf(argv[1],"%d",&N);

   sscanf(argv[2],"%d",&steps);
   printf ("N = %d steps = %d \n",N,steps);

   if (argc == 4) {
      sscanf(argv[3],"%s",file);
   }
   printf("Output will be dumped to <%s> \n\n",file);

   NN = N+2; // size of expanded matrix that includes boundary conditions
   size = NN*NN; // size = (N+2)^2 leaves room for boundary conditions

   if ((u = (float *) malloc(size*sizeof(float)))==NULL) {
      printf("Malloc Error: Memory Allocation Problem for size N=%d\n",N);
      exit(1);
   }
   if ((uold = (float *) malloc(size*sizeof(float)))==NULL) {
      printf("Malloc Error: Memory Allocation Problem for size N=%d\n",N);
      exit(1);
   }

   // Start timer
   startTimer(&timer_val);

   h = 1/(float) N;
   dt =  h/(float) sqrt((double)2.0);
   rho = dt/h;

   // create a disturbance in the middle of the wave plane
   for (i=0; i<NN; i++) {
      for (j=0; j<NN; j++) { // set background to
         u[i*NN+j]=0.0;        // ref level 0.0
      }
   }

   for (i=NN/2-1;i<=NN/2;i++) {
      for (j=NN/2-1;j<=NN/2;j++) { // depress middle 4
         u[i*NN+j] = -1.0;        // elements to -1
      }
   }

   // excite the region of interest
   u[(NN/2-2)*NN+NN/2-2] = 1.2; // raise up a 
   u[(NN/2-2)*NN+NN/2-1] = 1.4; // volumetrically equivalent
   u[(NN/2-2)*NN+NN/2]   = 1.4; // crest along the edges
   u[(NN/2-2)*NN+NN/2+1] = 1.2; // of the depressed region

   u[(NN/2+1)*NN+NN/2-2] = 1.4;
   u[(NN/2+1)*NN+NN/2+1] = 1.4;
   u[(NN/2)*NN+NN/2-2]   = 1.4;
   u[(NN/2)*NN+NN/2+1]   = 1.4;

   u[(NN/2+1)*NN+NN/2-2] = 1.2;
   u[(NN/2+1)*NN+NN/2+1] = 1.4;
   u[(NN/2+1)*NN+NN/2-2]   = 1.4;
   u[(NN/2+1)*NN+NN/2+1] = 1.2;

   // set up same conditions one time step earlier -- deriviative 0
   for (i=0; i<NN; i++) {
      for (j=0; j<NN; j++) {
         uold[i*NN+j]=u[i*NN+j];
      }
   }

   rho2=rho*rho;

   float *uoldd,*ud,*unewd;

   // Allocate device memory and Transfer host arrays u and uold 
   error_id=hipMalloc((void **) &uoldd, size*sizeof(float));
   if (error_id != hipSuccess) {
      printf( "Device Memory allocation for uoldd failed--returned %d\n-> %s\n",
          (int)error_id, hipGetErrorString(error_id) );
      exit(EXIT_FAILURE);
   }

   error_id=hipMemcpy(uoldd, uold, size*sizeof(float), hipMemcpyHostToDevice);
   if (error_id != hipSuccess) {
      printf( "Memory Copy for uoldd failed--returned %d\n-> %s\n",
          (int)error_id, hipGetErrorString(error_id) );
      exit(EXIT_FAILURE);
   }

   error_id=hipMalloc((void **) &ud, size*sizeof(float));
   if (error_id != hipSuccess) {
      printf( "Device Memory allocation for ud failed--returned %d\n-> %s\n",
          (int)error_id, hipGetErrorString(error_id) );
      exit(EXIT_FAILURE);
   }

   error_id=hipMemcpy(ud, u, size*sizeof(float), hipMemcpyHostToDevice);
   if (error_id != hipSuccess) {
      printf( "Memory Copy for ud failed--returned %d\n-> %s\n",
          (int)error_id, hipGetErrorString(error_id) );
      exit(EXIT_FAILURE);
   }

   // Allocate device memory of unew array for results
   // (note memory is copied because boundary conditions will be needed
   //  later when pointers are swapped)
   error_id=hipMalloc((void **) &unewd, size*sizeof(float));
   if (error_id != hipSuccess) {
      printf( "Device Memory allocation for unewd failed--returned %d\n-> %s\n",
          (int)error_id, hipGetErrorString(error_id) );
      exit(EXIT_FAILURE);
   }

   error_id=hipMemcpy(unewd, u, size*sizeof(float), hipMemcpyHostToDevice);
   if (error_id != hipSuccess) {
      printf( "Memory Copy for unewd failed--returned %d\n-> %s\n",
          (int)error_id, hipGetErrorString(error_id) );
      exit(EXIT_FAILURE);
   }


   // Setup the kernel execution configuration parameters
   dim3 dimGrid;
   dimGrid.x = iceil(N,TILE_WIDTH);
   dimGrid.y = iceil(N,TILE_WIDTH);
   dim3 dimBlock(TILE_WIDTH,TILE_WIDTH);

   for (t=0; t<steps; t++) {

      // Launch the kernel!!!
      WaveKernel<<<dimGrid, dimBlock>>>(unewd, ud, uoldd, NN, rho2);

      error_id=hipGetLastError();
      if (error_id != hipSuccess) {
         printf( "Attempted Launch of WaveKernel returned %d\n-> %s\n",
             (int)error_id, hipGetErrorString(error_id) );
         exit(EXIT_FAILURE);
      }

      // swap device pointers instead of moving data
      tmp_ptr = uoldd;
      uoldd    = ud;
      ud       = unewd;
      unewd    = tmp_ptr;
   }

   // Transfer P from device to host
   error_id=hipMemcpy(u,unewd,size*sizeof(float) ,hipMemcpyDeviceToHost);
   if (error_id != hipSuccess) {
      printf( "Memory Copy back to host for u failed--returned %d\n-> %s\n",
          (int)error_id, hipGetErrorString(error_id) );
      exit(EXIT_FAILURE);
   }


   // Free device matrices
   error_id=hipFree(uoldd);
   if (error_id != hipSuccess) {
      printf( "Cuda could not free memory uoldd -- returned %d\n-> %s\n",
          (int)error_id, hipGetErrorString(error_id) );
      exit(EXIT_FAILURE);
   }

   error_id=hipFree(ud);
   if (error_id != hipSuccess) {
      printf( "Cuda could not free memory ud -- returned %d\n-> %s\n",
          (int)error_id, hipGetErrorString(error_id) );
      exit(EXIT_FAILURE);
   }

   error_id=hipFree(unewd);
   if (error_id != hipSuccess) {
      printf( "Cuda could not free memory unewd -- returned %d\n-> %s\n",
          (int)error_id, hipGetErrorString(error_id) );
      exit(EXIT_FAILURE);
   }


   // End timer
   timer_val=stopNreadTimer(&timer_val);
   printf("Processing time : %f (ms)\n",timer_val*1000.);

   output_wave_gnu(NN,u,file);

}

